#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Arrhenius reaction solver functions
 *
*/
/** \file
 * \brief Arrhenius reaction solver functions
*/
extern "C"{
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "../rxns2.h"

#define TEMPERATURE_K_ env_data[0]
#define PRESSURE_PA_ env_data[1]

#define NUM_REACT_ int_data[0*n_rxn]
#define NUM_PROD_ int_data[1*n_rxn]
#define A_ float_data[0*n_rxn]
#define B_ float_data[1*n_rxn]
#define C_ float_data[2*n_rxn]
#define D_ float_data[3*n_rxn]
#define E_ float_data[4*n_rxn]
#define CONV_ float_data[5*n_rxn]
#define RATE_CONSTANT_ rxn_env_data[0*n_rxn]
#define NUM_INT_PROP_ 2
#define NUM_FLOAT_PROP_ 6
#define NUM_ENV_PARAM_ 1
#define REACT_(x) (int_data[(NUM_INT_PROP_ + x)*n_rxn]-1)
#define PROD_(x) (int_data[(NUM_INT_PROP_ + NUM_REACT_ + x)*n_rxn]-1)
#define DERIV_ID_(x) int_data[(NUM_INT_PROP_ + NUM_REACT_ + NUM_PROD_ + x)*n_rxn]
#define JAC_ID_(x) int_data[(NUM_INT_PROP_ + 2*(NUM_REACT_+NUM_PROD_) + x)*n_rxn]
#define YIELD_(x) float_data[(NUM_FLOAT_PROP_ + x)*n_rxn]
#define INT_DATA_SIZE_ (NUM_INT_PROP_+(NUM_REACT_+2)*(NUM_REACT_+NUM_PROD_))
#define FLOAT_DATA_SIZE_ (NUM_FLOAT_PROP_+NUM_PROD_)

/** \brief Calculate contributions to the time derivative \f$f(t,y)\f$ from
 * this reaction.
 *
 * \param model_data Pointer to the model data, including the state array
 * \param deriv Pointer to the time derivative to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being computed (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
#ifdef PMC_USE_SUNDIALS
#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void rxn2_arrhenius_calc_deriv_contrib(ModelData *model_data, realtype *deriv,
                                      int *rxn_int_data, double *rxn_float_data,
                                      double *rxn_env_data, double time_step)
{
#ifdef __CUDA_ARCH__
  int n_rxn=model_data->n_rxn;
#else
  int n_rxn=1;
#endif
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double *env_data = model_data->grid_cell_env;

  double rate = RATE_CONSTANT_;
  for (int i_spec=0; i_spec<NUM_REACT_; i_spec++)
    rate *= state[REACT_(i_spec)];

  // Add contributions to the time derivative
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<NUM_REACT_; i_spec++, i_dep_var++) {
      if (DERIV_ID_(i_dep_var) < 0) continue;
#ifdef __CUDA_ARCH__
        atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),-rate);
#else
        deriv[DERIV_ID_(i_dep_var)] -= rate;
#endif
	}
    for (int i_spec=0; i_spec<NUM_PROD_; i_spec++, i_dep_var++) {
      if (DERIV_ID_(i_dep_var) < 0) continue;

      // Negative yields are allowed, but prevented from causing negative
      // concentrations that lead to solver failures
      if (-rate*YIELD_(i_spec)*time_step <= state[PROD_(i_spec)]) {
#ifdef __CUDA_ARCH__
        atomicAdd(&(deriv[DERIV_ID_(i_dep_var)]),rate*YIELD_(i_spec));
#else
        deriv[DERIV_ID_(i_dep_var)] += rate * YIELD_(i_spec);
#endif
      }
    }
  }

}
#endif

/** \brief Calculate contributions to the Jacobian from this reaction
 *
 * \param model_data Pointer to the model data
 * \param J Pointer to the sparse Jacobian matrix to add contributions to
 * \param rxn_data Pointer to the reaction data
 * \param time_step Current time step being calculated (s)
 * \return The rxn_data pointer advanced by the size of the reaction data
 */
/*
#ifdef PMC_USE_SUNDIALS
#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void rxn_gpu_arrhenius_calc_jac_contrib(ModelData *model_data, realtype *J, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step)
{
#ifdef __CUDA_ARCH__
  int n_rxn=model_data->n_rxn;
#else
  int n_rxn=1;;
#endif
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double *env_data = model_data->grid_cell_env;

  // Add contributions to the Jacobian
  int i_elem = 0;
  for (int i_ind = 0; i_ind < NUM_REACT_; i_ind++) {
    // Calculate d_rate / d_i_ind
    realtype rate = RATE_CONSTANT_;
    for (int i_spec = 0; i_spec < NUM_REACT_; i_spec++)
      if (i_spec != i_ind) rate *= state[REACT_(i_spec)];

    for (int i_dep=0; i_dep<NUM_REACT_; i_dep++, i_elem++) {
      if (JAC_ID_(i_elem) < 0) continue;
#ifdef __CUDA_ARCH__
      atomicAdd(&(J[JAC_ID_(i_elem)]),-rate);
#else
      J[JAC_ID_(i_elem)] -= rate;
#endif
    }
    for (int i_dep=0; i_dep<NUM_PROD_; i_dep++, i_elem++) {
  if (JAC_ID_(i_elem) < 0) continue;
      // Negative yields are allowed, but prevented from causing negative
      // concentrations that lead to solver failures
      if (-rate * state[REACT_(i_ind)] * YIELD_(i_dep) * time_step <= state[PROD_(i_dep)]) {
#ifdef __CUDA_ARCH__
    atomicAdd(&(J[JAC_ID_(i_elem)]),YIELD_(i_dep) * rate);
#else
    J[JAC_ID_(i_elem)] += YIELD_(i_dep) * rate;
#endif
      }
    }
  }

}
#endif
*/


}
